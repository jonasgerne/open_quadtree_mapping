#include "hip/hip_runtime.h"
#include <quadmap/depth_fusion.cuh>

#ifndef M_PI
#define M_PI       3.14159265358979323846   // pi
#endif

namespace quadmap
{

__global__ void high_gradient_filter
(DeviceImage<float> *depth_output_devptr,
  DeviceImage<float> *filtered_depth_devptr)
{
  const int x = threadIdx.x + blockIdx.x * blockDim.x;
  const int y = threadIdx.y + blockIdx.y * blockDim.y;
  const int width = depth_output_devptr->width;
  const int height = depth_output_devptr->height;

  if(x >= width - 1 || y >= height - 1 || x == 0 || y == 0)
    return;

  float gradient_x = depth_output_devptr->atXY(x+1,y)-depth_output_devptr->atXY(x-1,y);
  float gradient_y = depth_output_devptr->atXY(x,y+1)-depth_output_devptr->atXY(x,y-1);
  if(gradient_x*gradient_x + gradient_y*gradient_y > 0.01)
    filtered_depth_devptr->atXY(x,y) = -1;
}

__global__ void fuse_transform(
  DeviceImage<float4> *pre_seeds_devptr,
  DeviceImage<int> *transform_table_devptr,
  SE3<float> last_to_cur,
  PinholeCamera camera,
  const float min_inlier_ratio_bad)
{
  const int x = threadIdx.x + blockIdx.x * blockDim.x;
  const int y = threadIdx.y + blockIdx.y * blockDim.y;
  const int width = pre_seeds_devptr->width;
  const int height = pre_seeds_devptr->height;

  if(x >= width || y >= height)
    return;
  
  const int index = x + y * width;

  const float3 dir = normalize(camera.cam2world(make_float2(x, y)));

  float4 pixel_info = pre_seeds_devptr->atXY(x,y);

  if ( is_badpoint(pixel_info, min_inlier_ratio_bad) )
    return;

  // Transform point from last to current frame
  float3 projected = last_to_cur * (dir * pixel_info.z);

  // Ignore if closer than min depth
  float new_depth = length(projected);
  if(new_depth <= MIN_DEP)
    return;

  pixel_info.z = new_depth;
  // Accumulate variance ??
  // pixel_info.w += new_depth * 0.01; // TODO: Add parameter
  // pixel_info.y *= 1.001;

  // Project onto current image
  const float2 project_point = camera.world2cam(projected);
  const int projecte_x = project_point.x + 0.5;
  const int projecte_y = project_point.y + 0.5;

  //projected out of the image
  if(projecte_x >= width || projecte_x < 0 || projecte_y >= height || projecte_y < 0)
    return;

  //check color diff
  // float origin_color = tex2D(pre_image_tex, x + 0.5, y + 0.5);
  // float trans_color = tex2D(income_image_tex, projecte_x + 0.5, projecte_y + 0.5);
  // if( fabs(origin_color-trans_color) > 30.0 )
  //   return;

  // Depth map culling over multiple threads
  int *check_ptr = &(transform_table_devptr->atXY(projecte_x, projecte_y));
  int expect_i = 0;
  int actual_i;
  bool finish_job = false;
  int max_loop = 5;
  while(!finish_job && max_loop > 0)
  {
    max_loop--;
    actual_i = atomicCAS(check_ptr, expect_i, index);
    if(actual_i != expect_i)
    {
      int now_x = actual_i % width;
      int now_y = actual_i / width;
      float now_d = (pre_seeds_devptr->atXY(now_x, now_y)).z;
      if(now_d < new_depth)
        finish_job = true;
    }
    else
    {
      finish_job = true;
    }
    expect_i = actual_i;
  }

  pre_seeds_devptr->atXY(x,y) = pixel_info;
}

__global__ void hole_filling(DeviceImage<int> *transform_table_devptr)
{
  const int x = threadIdx.x + blockIdx.x * blockDim.x;
  const int y = threadIdx.y + blockIdx.y * blockDim.y;
  const int width = transform_table_devptr->width;
  const int height = transform_table_devptr->height;

  if(x >= width - 1 || y >= height - 1 || x <= 1 || y <= 1)
    return;

  const int transform_i = transform_table_devptr->atXY(x,y);

  if(transform_i == 0)
    return;

  // Propagate depth to neighbors, if they are not set
  for(int i = -1; i <= 1; i++)
  {
    for(int j = -1; j <= 1; j++)
    {
      int *neighbor = &(transform_table_devptr->atXY(x + j, y + i));
      atomicCAS(neighbor, 0, transform_i);
    }
  }
}

__global__ void fuse_currentmap(
  DeviceImage<int> *transform_table_devptr,
  DeviceImage<float> *depth_output_devptr,
  DeviceImage<float4> *former_depth_devptr,
  DeviceImage<float4> *new_depth_devptr,
  const float min_inlier_ratio_good,
  const float new_variance_factor,
  const float prev_variance_factor,
  const float variance_offset)
{
  const int x = threadIdx.x + blockIdx.x * blockDim.x;
  const int y = threadIdx.y + blockIdx.y * blockDim.y;

  const int width = transform_table_devptr->width;
  const int height = transform_table_devptr->height;

  if (x >= width || y >= height)
    return;

  // Current measurement and uncertainty
  float depth_estimate = depth_output_devptr->atXY(x,y);
  // float uncertianity = depth_estimate * depth_estimate * 0.01;
  // float uncertianity = 1.0;
  float uncertianity = fmaxf(0.5,depth_estimate*0.2); //TODO: Add parameter
  uncertianity *= uncertianity;
  if (depth_estimate <= 0.0f)
    uncertianity = 1e9;

  //printf("Uncertainty %f\n", uncertianity);

  // Get previous estimate
  int pre_position = transform_table_devptr->atXY(x,y);
  float4 pixel_info;
  if (pre_position > 0)
    pixel_info = former_depth_devptr->atXY(pre_position%width, pre_position/width);
  else
  {
    // Assume initial estimate with current depth
    pixel_info = make_float4(initial_a, initial_b, depth_estimate, initial_variance);
  }

  // Reset previous estimate if depth difference is bigger than sum of variances, eg due to an occlusion
  if ( (depth_estimate - pixel_info.z)*(depth_estimate - pixel_info.z) > uncertianity * new_variance_factor + pixel_info.w * prev_variance_factor + variance_offset)
     pixel_info = make_float4(initial_a, initial_b, depth_estimate, initial_variance);

  // Previous estimate
  float a = pixel_info.x;
  float b = pixel_info.y;
  float miu = pixel_info.z;
  float sigma_sq = pixel_info.w;

  // Update based on variance
  float new_sq = uncertianity * sigma_sq / (uncertianity + sigma_sq);
  float new_miu = (depth_estimate * sigma_sq + miu * uncertianity) / (uncertianity + sigma_sq);
  float c1 = (a / (a+b)) * normpdf(depth_estimate, miu, uncertianity + sigma_sq);
  float c2 = (b / (a+b)) * 1 / 50.0f; //TODO: Add parameter

  // Update based on outlier ratio
  const float norm_const = c1 + c2;
  c1 = c1 / norm_const;
  c2 = c2 / norm_const;
  const float f = c1 * ((a + 1.0f) / (a + b + 1.0f)) + c2 *(a / (a + b + 1.0f));
  const float e = c1 * (( (a + 1.0f)*(a + 2.0f)) / ((a + b + 1.0f) * (a + b + 2.0f))) +
                  c2 *(a*(a + 1.0f) / ((a + b + 1.0f) * (a + b + 2.0f)));

  const float mu_prime = c1 * new_miu + c2 * miu;
  const float sigma_prime = c1 * (new_sq + new_miu * new_miu) + c2 * (sigma_sq + miu * miu) - mu_prime * mu_prime;
  const float a_prime = ( e - f ) / ( f - e/f );
  const float b_prime = a_prime * ( 1.0f - f ) / f;
  const float4 updated = make_float4(a_prime, b_prime, mu_prime, sigma_prime);

  __syncthreads();

//  depth_output_devptr->atXY(x,y) = mu_prime;
//  return;

  // (point_info.x /(point_info.x + point_info.y) > 0.60)
  //if(pre_position > 0)
//  if (pixel_info.x /(pixel_info.x + pixel_info.y) > 0.6) // inlier ratio: a / (a + b)
//    printf("%f / (%f + %f) = %f > 0.6\n", pixel_info.x, pixel_info.x, pixel_info.y, pixel_info.x /(pixel_info.x + pixel_info.y));

  // Check previous inlier ratio
  if(is_goodpoint(pixel_info, min_inlier_ratio_good))
    depth_output_devptr->atXY(x,y) = mu_prime;
  else
    depth_output_devptr->atXY(x,y) = -1.0f;

  new_depth_devptr->atXY(x,y) = updated;
}

}//namespace